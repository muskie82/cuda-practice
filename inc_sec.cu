#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N (32)
__global__ void inc(int *array, int len)
{
    int i;
    for (i = 0; i < len; i++)
        array[i]++;
    return;
}

int main(int argc, char *argv[])
{
    int i;
    int arrayH[N];
    int *arrayD;
    size_t array_size;
    for (i=0; i<N; i++) arrayH[i] = i;
    printf("input: ");
    for (i=0; i<N; i++)
        printf("%d ", arrayH[i]);
    printf("\n");
    array_size = sizeof(int) * N;
    hipMalloc((void **)&arrayD, array_size);
    hipMemcpy(arrayD, arrayH, array_size,
               hipMemcpyHostToDevice);
    inc<<<1, 1>>>(arrayD, N);
    hipMemcpy(arrayH, arrayD, array_size,
               hipMemcpyDeviceToHost);
    printf("output: ");
    for (i=0; i<N; i++)
        printf("%d ", arrayH[i]);
    printf("\n");
    return 0;
}