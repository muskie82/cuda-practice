#include "hip/hip_runtime.h"
#include<opencv2/opencv.hpp>
#include<iostream>

__global__ void convert2gray(uchar3 *color_pixel, unsigned char* gray_pixel){
    int ID = blockIdx.x*blockDim.x+threadIdx.x;

    gray_pixel[ID] = (unsigned char)(0.299f*color_pixel[ID].x
            + 0.586f*(float)color_pixel[ID].y
            + 0.114f*(float)color_pixel[ID].z);
}

int main(){
    // nvcc grayscale.cu -lopencv_core -lopencv_highgui -lopencv_imgproc -lopencv_imgcodecs
    cv::Mat input_img = cv::imread("dear.jpg",1);
    if(input_img.empty()==true){
        return 1;
    }

    int width = input_img.cols;
    int height = input_img.rows;
    std::cout << "Image_size : " << width << "×" << height << std::endl;

    uchar3* host_img_array_color = new uchar3[width*height];
    unsigned char* host_img_array_gray = new unsigned char[width*height];

    for(int y=0; y<height; y++){
        for(int x=0; x<width; x++){
            host_img_array_color[x+y*width]
            = make_uchar3(input_img.at<cv::Vec3b>(y, x)[2], input_img.at<cv::Vec3b>(y, x)[1], input_img.at<cv::Vec3b>(y, x)[0]);
        }
    }

    uchar3* device_img_array_color;
    unsigned char* device_img_array_gray;
    int datasize_color = sizeof(uchar3) * width * height;
    int datasize_gray = sizeof(unsigned char) * width * height;
    hipMalloc((void**)&device_img_array_color, datasize_color);
    hipMalloc((void**)&device_img_array_gray, datasize_gray);

    hipMemcpy(device_img_array_color, host_img_array_color, datasize_color, hipMemcpyHostToDevice);

    convert2gray<<<width*height,1>>> (device_img_array_color, device_img_array_gray);

    hipMemcpy(host_img_array_gray, device_img_array_gray, datasize_gray, hipMemcpyDeviceToHost);

    cv::Mat1b output_img(height, width);
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            output_img.at<unsigned char>(y, x) = host_img_array_gray[x + y * width];
        }
    }
    cv::imwrite("gray.jpg", output_img);

    hipFree(device_img_array_color);
    hipFree(device_img_array_gray);
    delete host_img_array_color;
    delete host_img_array_gray;

    return 0;
}