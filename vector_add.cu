#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n){
	for(int i=0;i<n;i++){
		out[i]=a[i]+b[i];
	}
}


int main(){
	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	a=(float*)malloc(sizeof(float)*N);
        b=(float*)malloc(sizeof(float)*N);
        out=(float*)malloc(sizeof(float)*N);

	for(int i=0; i<N; i++){
		a[i]=1.0f; b[i]=2.0f;
	}

	hipMalloc((void**)&d_a,sizeof(float)*N);
        hipMalloc((void**)&d_b,sizeof(float)*N);
        hipMalloc((void**)&d_out,sizeof(float)*N);

	hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

	vector_add<<<1,1>>>(d_out, d_a, d_b, N);

	hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    printf("%f\n", out[0]);
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    // Deallocate host memory
    free(a);
    free(b);
    free(out);

	return 0;
}
